#include "hip/hip_runtime.h"
#include "relu.hpp"
// #include "tensor.hpp"
#include <glog/logging.h>


#define eee(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void relu_kernel(float *input, float *output, uint thread_PerBlock, uint length) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y  + threadIdx.y;
    if(y*thread_PerBlock+x >= length) {
        return;
    }

    if(input[y*thread_PerBlock + x] >=0)
    {
        output[y*thread_PerBlock + x] = input[y*thread_PerBlock+x];
    } else {
        output[y*thread_PerBlock + x] = 0;
    }
}

void relu_layer::forward(float *input, float *output){
    CHECK(m_length != 0) << "m_length is 0";

    float* d_input;
    eee(hipMalloc((void**)&d_input, sizeof(float) * m_length));
    float* d_output;
    eee(hipMalloc((void**)&d_output, sizeof(float) * m_length));

    eee(hipMemcpy(d_input, input, m_length*sizeof(float), hipMemcpyHostToDevice));

    uint thread_PerBlock = 32;
    dim3 rowsGrid(1, ceil(1.0f*m_length/thread_PerBlock), 1);
	dim3 rowsThreads(thread_PerBlock, 1, 1);

    relu_kernel<<<rowsGrid, rowsThreads>>>(d_input, d_output, thread_PerBlock, m_length);

    eee(hipMemcpy(output, d_output, m_length * sizeof(float), hipMemcpyDeviceToHost));

}