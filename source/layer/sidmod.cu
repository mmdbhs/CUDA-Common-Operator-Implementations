#include "hip/hip_runtime.h"
#include "sigmod.hpp"
#include <glog/logging.h>


#define eee(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void sigmod_kernel(float *input, float *output, uint length) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
	// uint y = blockIdx.y * blockDim.y  + threadIdx.y;
    if(x >= length) {
        return;
    }

    output[x] = 1.0 / (1.0 + expf(input[x]));
}

void sigmod_layer::forward(float *input, float *output){
    CHECK(m_length != 0) << "m_length is 0";

    float* d_input;
    eee(hipMalloc((void**)&d_input, sizeof(float) * m_length));
    float* d_output;
    eee(hipMalloc((void**)&d_output, sizeof(float) * m_length));

    eee(hipMemcpy(d_input, input, m_length*sizeof(float), hipMemcpyHostToDevice));

    uint thread_PerBlock = 32;
    dim3 rowsGrid(ceil(1.0f*m_length/thread_PerBlock),1 , 1);
	dim3 rowsThreads(thread_PerBlock, 1, 1);

    sigmod_kernel<<<rowsGrid, rowsThreads>>>(d_input, d_output, m_length);

    eee(hipMemcpy(output, d_output, m_length * sizeof(float), hipMemcpyDeviceToHost));

}